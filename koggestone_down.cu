/* -*- mode: c++ -*- */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>

void checkCuda(const hipError_t s, const char* file, const int line) {
  if(s != hipSuccess) {
    fprintf(stderr, 
	    "%s:%d: CUDA error: %s\n", file, line, hipGetErrorString(s));
    exit(1);
  }
}

#define check_cuda(x) checkCuda((x), __FILE__, __LINE__)


__global__
void sum_warp(int *n, int N, int *out) {
  int tid = threadIdx.x;
  int temp1 = n[tid];
  int temp2;

  assert(N <= 32);
    for(int d = 0; d < 5; d++) {        
   	temp2 = __shfl_down(temp1,((1<<d))); 
   if((tid % (1<<(d+1))) == 0) {
	temp1+=temp2;
    }
  } 

  if(tid == 0) {
    *out = temp1;
  }  

}

int main(int argc, char *argv[])
{
  if(argc == 1) {
    fprintf(stderr, "Usage: %s number1 number2...\n", argv[0]);
    exit(1);
  }

  if(argc > 33) {
    fprintf(stderr, "Usage: %s number1 number2...\n", argv[0]);
    fprintf(stderr, "Can only add up to 32 numbers\n");
    exit(1);
  }

  int n[32], N = 0;

  for(int i = 0; i < argc-1; i++) {
    n[i] = atoi(argv[i+1]);
    N++;
  }

  printf("Read %d numbers.\n", N);

  int *n_d, *out_d;
  int out;
  
  check_cuda(hipMalloc(&n_d, sizeof(int) * N));
  check_cuda(hipMalloc(&out_d, sizeof(int) * 1));
  
  check_cuda(hipMemcpy(n_d, n, sizeof(int) * N, hipMemcpyHostToDevice));
  
  sum_warp<<<1, 32>>>(n_d, N, out_d);

  check_cuda(hipMemcpy(&out, out_d, sizeof(int) * 1, hipMemcpyDeviceToHost));

  printf("Sum of %d numbers: %d\n", N, out);

  check_cuda(hipFree(n_d));
  check_cuda(hipFree(out_d));

  return 0;
}
